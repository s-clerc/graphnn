#include "hip/hip_runtime.h"
#include "tensor/gpu_handle.h"
#include "util/gnn_macros.h"
#include "util/mem_holder.h"
#include "tbb/tbb.h"

namespace gnn
{

__global__ void SetupRandKernel(hiprandState_t *state, unsigned long long seed) 
{
    const unsigned int tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    /* Each thread gets same seed, a different sequence number,
     no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}

void GpuHandle::Init(int dev_id, unsigned int _streamcnt)
{
	// My understanding is that according to https://github.com/DrTimothyAldenDavis/SuiteSparse/issues/72
	// in modern oneTBB, excluding the line below means that the system will automatically allocate threads
	// and since it is fixed to a constant, I don't think it does anything very important wrt to the other
	// libraries being used.
	// If needed, we could use https://oneapi-src.github.io/oneTBB/main/tbb_userguide/Migration_Guide/Task_Scheduler_Init.html
	// 's max concurrency to limit it to 4 I believe.
	//tbb::task_scheduler_init init(4);
	streamcnt = _streamcnt;
	hipDeviceReset();
	hipSetDevice(dev_id);

	cublashandles = new hipblasHandle_t[streamcnt];
	cusparsehandles = new hipsparseHandle_t[streamcnt];
	inUse = new bool[streamcnt];
	while (!resources.empty())
		resources.pop();
	for (unsigned int id = 0; id < streamcnt; ++id)
	{
		hipblasCreate(&cublashandles[id]);	
		hipsparseCreate(&cusparsehandles[id]);
		inUse[id] = false;
		resources.push(id);
	}
	hipStreamCreate(&cudaRandStream);

	hiprandCreateGenerator(&curandgenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
	
	hiprandSetPseudoRandomGeneratorSeed(curandgenerator, time(NULL));
	
    hipMalloc((void **)&devRandStates, NUM_RND_STREAMS * sizeof(hiprandState_t));
	SetupRandKernel<<<NUM_RND_BLOCKS, NUM_RND_THREADS_PER_BLOCK>>>(devRandStates, 1 + time(NULL)*2);
}

GpuContext GpuHandle::AquireCtx()
{
	r_loc.lock();
	ASSERT(resources.size(), "running out of gpu resources");

	int cur_pos = resources.front();
	resources.pop();

	r_loc.unlock();

	ASSERT(!inUse[cur_pos], "logic error: in-use resource is found available");
	inUse[cur_pos] = true;	
	hipblasSetStream(cublashandles[cur_pos], hipStreamPerThread);
	hipsparseSetStream(cusparsehandles[cur_pos], hipStreamPerThread);
	return GpuContext(cur_pos, cublashandles[cur_pos], cusparsehandles[cur_pos]);
}

void GpuHandle::ReleaseCtx(const GpuContext& ctx)
{
	r_loc.lock();
	resources.push(ctx.id);
	ASSERT(inUse[ctx.id], "logic error: in-use resource is not recorded, or you are releasing same resource multiple times");
	inUse[ctx.id] = false;
	r_loc.unlock();	
}

void GpuHandle::Destroy()
{
	hipDeviceSynchronize();
	hipStreamDestroy(cudaRandStream);
	for (unsigned int id = 0; id < streamcnt; ++id)
	{
		hipblasDestroy(cublashandles[id]);
		hipsparseDestroy(cusparsehandles[id]);
	}
	delete[] cublashandles;
	delete[] cusparsehandles;
	delete[] inUse;
	hiprandDestroyGenerator(curandgenerator);
    hipFree(devRandStates);
	streamcnt = 0U;
}

hiprandState_t* GpuHandle::devRandStates = NULL;
hipblasHandle_t* GpuHandle::cublashandles = NULL;
hipsparseHandle_t* GpuHandle::cusparsehandles = NULL;
hiprandGenerator_t GpuHandle::curandgenerator;
unsigned int GpuHandle::streamcnt = 1U;
std::queue< int > GpuHandle::resources;
std::mutex GpuHandle::r_loc;
std::mutex GpuHandle::rand_lock;
bool* GpuHandle::inUse = NULL;
hipStream_t GpuHandle::cudaRandStream;

}